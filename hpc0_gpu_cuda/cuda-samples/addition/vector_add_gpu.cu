// SDSC Summer Institute 2015
// Andreas Goetz (agoetz@sdsc.edu)

// CUDA program to add two vectors in parallel on the GPU
// launch all kernels at once
//


#include <hip/hip_runtime.h>
#include<stdio.h>

// define vector length and threads per block
#define N (255*2047)
#define TPB 512

//
// CUDA device function that adds two integer vectors
//
__global__ void add(int *a, int *b, int *c, int n){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < n)
    c[tid] = a[tid] + b[tid];

}

//
// main program
//
int main(void){

  int h_a[N], h_b[N], h_c[N];
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);
  int i, nblock, err;

  // allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize vectors
  for (i=0; i<N; i++){
    h_a[i] = i+1;
    h_b[i] = i+1;
  }

  // copy input data to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // add vectors by launching a sufficient number of blocks of the add() kernel
  nblock = (N+TPB-1)/TPB;
  printf("\nLaunching vector addition kernel...\n");
  printf("Vector length     = %d\n",N);
  printf("Blocks            = %d\n",nblock);
  printf("Threads per block = %d\n",TPB);
  printf("Kernel copies     = %d\n",nblock*TPB);
  add<<<nblock,TPB>>>(d_a, d_b, d_c, N);

  // copy results back to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // check results
  err = 0;
  for (i=0; i<N; i++){
    if (h_c[i] != 2*(i+1)) err = 1;
  }
  if (err != 0){
    printf("\n Error, %d elements do not match!\n\n", err);
  } else {
    printf("\n Success! All elements match.\n\n");
  }

  return 0;

}
